#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "BMP.h"

enum Color{Red, Blue, Green};

BMP persistColor(BMP *bmp, Color color);

__global__ void PersistColor(BMP *bmp, Color color)
{
	// The goal of this kernel is to remove all colors except for black, white, and the designated color (red, blue, or green)
}

int main()
{
	BMP bmp;
	bmp.LoadFromFile("cat.bmp");
	bmp.OutputFileData();

	BMP outputBMP = persistColor(&bmp, Blue);

	hipError_t cudaStatus;
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
	    fprintf(stderr, "hipDeviceReset failed!");
	    return 1;
	}


	printf("\nProgram completed.\n");
	std::getchar();
	return 0;
  
}

BMP persistColor(BMP *bmp, Color color)
{
	//TODO
	// 1. Add Timing
	// 
	BMP *dev_bmp;
	BMP *host_output_bmp;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipHostMalloc((void**)&host_output_bmp, sizeof(BMP));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipHostMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_bmp, sizeof(BMP));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_bmp, bmp, sizeof(BMP), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	PersistColor<<<1, sizeof(BMP) >>>(dev_bmp, color);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "PersistColor launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(host_output_bmp, dev_bmp, sizeof(BMP), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_bmp);
	return *host_output_bmp;
}

